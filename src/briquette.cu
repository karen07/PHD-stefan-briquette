
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdlib.h>
#include <string>

using namespace std;

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__device__ double delt_d;
__device__ double freeze_temp_d;

__device__ double initial_d;
__device__ double bound_d;

__device__ int problem_size_d;
__device__ double dx_d;
__device__ double dt_d;

__device__ int get_pos(int i, int j, int k)
{
    return i + j * problem_size_d + k * problem_size_d * problem_size_d;
}

__device__ double c_ro(double t)
{
    double c_frosen = 1600;
    double ro_frosen = 1200;

    double c_melt = 1900;
    double ro_melt = 1200;

    double L = 330000 * 0.1;

    if (t < freeze_temp_d - delt_d)
        return c_frosen * ro_frosen;

    if (t >= freeze_temp_d - delt_d && t < freeze_temp_d)
        return (c_frosen + L / delt_d / 2) * ro_frosen;

    if (t >= freeze_temp_d && t < freeze_temp_d + delt_d)
        return (c_melt + L / delt_d / 2) * ro_melt;

    if (t >= freeze_temp_d + delt_d)
        return c_melt * ro_melt;

    return 0;
}

__device__ double k(double t)
{
    double k_frosen = 0.92;

    double k_melt = 0.72;

    double scale = 3600;

    if (t < freeze_temp_d)
        return k_frosen * scale;

    if (t >= freeze_temp_d)
        return k_melt * scale;

    return 0;
}

__global__ void init(double *in)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int idz = threadIdx.z + blockDim.z * blockIdx.z;

    if (idx < problem_size_d && idy < problem_size_d && idz < problem_size_d) {
        if (idx > 0 && idx < problem_size_d - 1 && idy > 0 && idy < problem_size_d - 1 && idz > 0 &&
            idz < problem_size_d - 1) {
            in[get_pos(idx, idy, idz)] = initial_d;
        } else {
            in[get_pos(idx, idy, idz)] = bound_d;
        }
    }
}

__global__ void solve(double *out, double *in)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int idz = threadIdx.z + blockDim.z * blockIdx.z;

    if (idx > 0 && idx < problem_size_d - 1 && idy > 0 && idy < problem_size_d - 1 && idz > 0 &&
        idz < problem_size_d - 1)
        out[get_pos(idx, idy, idz)] =
            in[get_pos(idx, idy, idz)] +
            dt_d / ((dx_d * dx_d) * c_ro(in[get_pos(idx, idy, idz)])) *
                ((((k(in[get_pos(idx + 1, idy, idz)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx + 1, idy, idz)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx - 1, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx - 1, idy, idz)])) +
                 (((k(in[get_pos(idx, idy + 1, idz)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy + 1, idz)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx, idy - 1, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx, idy - 1, idz)])) +
                 (((k(in[get_pos(idx, idy, idz + 1)]) + k(in[get_pos(idx, idy, idz)])) / 2.0) *
                      (in[get_pos(idx, idy, idz + 1)] - in[get_pos(idx, idy, idz)]) -
                  ((k(in[get_pos(idx, idy, idz)]) + k(in[get_pos(idx, idy, idz - 1)])) / 2.0) *
                      (in[get_pos(idx, idy, idz)] - in[get_pos(idx, idy, idz - 1)])));
}

int main()
{
    int max_size = 300;

    double dt = 0.0001;
    double dx = 0.01;
    double delt = 0.1;

    double initial = 15;
    double freeze_temp = 0;

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &dt, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dx_d), &dx, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(initial_d), &initial, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(freeze_temp_d), &freeze_temp, sizeof(double)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(delt_d), &delt, sizeof(double)));

    double *heat_array_old;
    gpuErrchk(
        hipMalloc((void **)&heat_array_old, max_size * max_size * max_size * sizeof(double)));

    double *heat_array_now;
    gpuErrchk(
        hipMalloc((void **)&heat_array_now, max_size * max_size * max_size * sizeof(double)));

    double *heat_array_cpu = (double *)malloc(max_size * max_size * max_size * sizeof(double));

    ofstream file_out("out.txt");

    for (int j = -10; j <= -10; j += 5) {
        for (int i = 40; i <= 40; i += 5) {
            int problem_size = i * 0.01 / dx + 1;
            gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(problem_size_d), &problem_size, sizeof(int)));

            double bound_val = j;
            gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bound_d), &bound_val, sizeof(double)));

            dim3 threadsPerBlock(8, 8, 8);
            dim3 numBlocks(problem_size / threadsPerBlock.x + 1,
                           problem_size / threadsPerBlock.y + 1,
                           problem_size / threadsPerBlock.z + 1);

            file_out << "Air temperature:" << bound_val << "C "
                     << "Briquette side size:" << i * 0.01 << "m" << endl;
            file_out << "Time(hours)\tMaximum temperature of the briquette(C)" << endl;

            init<<<numBlocks, threadsPerBlock>>>(heat_array_old);
            init<<<numBlocks, threadsPerBlock>>>(heat_array_now);

            double centre = 0;

            for (int time = 0; time <= 8 / dt; time++) {
                solve<<<numBlocks, threadsPerBlock>>>(heat_array_now, heat_array_old);

                double *tmp = heat_array_now;
                heat_array_now = heat_array_old;
                heat_array_old = tmp;

                if (time % ((int)(1 / dt)) == 0) {
                    gpuErrchk(hipMemcpy(
                        &centre,
                        &heat_array_old[problem_size / 2 + problem_size / 2 * problem_size +
                                        problem_size / 2 * problem_size * problem_size],
                        sizeof(double), hipMemcpyDeviceToHost));
                    file_out << time * dt << "\t" << centre << endl;

                    gpuErrchk(
                        hipMemcpy(heat_array_cpu, heat_array_old,
                                   problem_size * problem_size * problem_size * sizeof(double),
                                   hipMemcpyDeviceToHost));
                    char out_string[100];
                    sprintf(out_string, "plot/result_%d.vtk", time);
                    ofstream out(out_string);
                    out << "# vtk DataFile Version 2.0" << endl;
                    out << "Heat" << endl;
                    out << "ASCII" << endl;
                    out << "DATASET STRUCTURED_POINTS" << endl;
                    out << "DIMENSIONS " << problem_size << " " << problem_size << " "
                        << problem_size << endl;
                    out << "ASPECT_RATIO 1 1 1" << endl;
                    out << "ORIGIN 0 0 0" << endl;
                    out << "POINT_DATA " << problem_size * problem_size * problem_size << endl;
                    out << "SCALARS heat float 1" << endl;
                    out << "LOOKUP_TABLE default" << endl;
                    for (int i = 0; i < problem_size; i++) {
                        for (int j = 0; j < problem_size; j++)
                            for (int k = 0; k < problem_size; k++) {
                                out << heat_array_cpu[i + j * problem_size +
                                                      k * problem_size * problem_size]
                                    << " ";
                            }
                        out << endl;
                    }
                    out.close();
                }
            }

            file_out << endl;
        }
    }

    return 0;
}
